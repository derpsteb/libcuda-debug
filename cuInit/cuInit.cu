#include <stdio.h>

#include <hip/hip_runtime.h>

/**
* Host main routine
*/
int main(void) {
  // Error code to check return values for CUDA calls
  hipError_t hipSuccess = hipSuccess;
  // int cuDevice = 0;
  // CUcontext *cuCtx = NULL;

  printf("cuInit\n");

  hipError_t err = hipInit(0);
  if (err != hipSuccess) {
    const char *ret = NULL;
    hipDrvGetErrorString(err, &ret);
    fprintf(stderr, "error: %s\n", ret);
    exit(EXIT_FAILURE);
  }

//   printf("cuDeviceGet\n");

//   err = cuDeviceGet(&cuDevice, 0);
//   if (err != cudaSuccess) {
//     const char *ret = NULL;
//     cuGetErrorString(err, &ret);
//     fprintf(stderr, "error: %s\n", ret);
//     exit(EXIT_FAILURE);
//   }

//   printf("cuDevicePrimaryCtxRetain\n");

//   err = cuDevicePrimaryCtxRetain(cuCtx, cuDevice);
//   if (err != cudaSuccess) {
//     const char *ret = NULL;
//     cuGetErrorString(err, &ret);
//     fprintf(stderr, "error: %s\n", ret);
//     exit(EXIT_FAILURE);
//   }


  printf("Done\n");
  return 0;
}
