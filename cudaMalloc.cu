#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

/**
* Host main routine
*/
int main(void) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  printf("calling hipMalloc\n");
  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, 50000 * sizeof(float));
  printf("hipMalloc returned\n");

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  printf("calling hipFree\n");
  // Free device global memory
  err = hipFree(d_A);
  printf("hipFree returned\n");

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }


  printf("Done\n");
  return 0;
}
