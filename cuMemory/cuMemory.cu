#include <stdio.h>

#include <hip/hip_runtime.h>

/**
* Host main routine
*/
int main(void) {
  // Error code to check return values for CUDA calls
  hipError_t hipSuccess = hipSuccess;
  hipDevice_t cuDevice;
  hipCtx_t cuContext;
  // hipDeviceptr_t devicePtr;

  printf("hipInit\n");

  hipError_t err = hipInit(0);
  if (err != hipSuccess) {
    const char *ret = NULL;
    hipDrvGetErrorString(err, &ret);
    fprintf(stderr, "error: %s\n", ret);
    exit(EXIT_FAILURE);
  }

  printf("hipDeviceGet\n");

  err = hipDeviceGet(&cuDevice, 0);
  if (err != hipSuccess) {
    const char *ret = NULL;
    hipDrvGetErrorString(err, &ret);
    fprintf(stderr, "error: %s\n", ret);
    exit(EXIT_FAILURE);
  }

  // printf("hipCtxGetCurrent\n");

  // err = hipCtxGetCurrent(&cuContext);
  // if (err != hipSuccess) {
  //   const char *ret = NULL;
  //   hipDrvGetErrorString(err, &ret);
  //   fprintf(stderr, "error: %s\n", ret);
  //   exit(EXIT_FAILURE);
  // }


  printf("hipCtxCreate\n");

  err = hipCtxCreate(&cuContext, 0, cuDevice);
  if (err != hipSuccess) {
    const char *ret = NULL;
    hipDrvGetErrorString(err, &ret);
    fprintf(stderr, "error: %s\n", ret);
    exit(EXIT_FAILURE);
  }

  // printf("hipDevicePrimaryCtxRetain\n");
  // hipDevicePrimaryCtxRetain(&cuContext, cuDevice);
  // if (err != hipSuccess) {
  //   const char *ret = NULL;
  //   hipDrvGetErrorString(err, &ret);
  //   fprintf(stderr, "error: %s\n", ret);
  //   exit(EXIT_FAILURE);
  // }
  // printf("hipMalloc\n");

  // err = hipMalloc(&devicePtr, 50000 * sizeof(float));
  // if (err != hipSuccess) {
  //   const char *ret = NULL;
  //   hipDrvGetErrorString(err, &ret);
  //   fprintf(stderr, "error: %s\n", ret);
  //   exit(EXIT_FAILURE);
  // }


  printf("Done\n");
  return 0;
}
